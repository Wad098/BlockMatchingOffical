#include "hip/hip_runtime.h"
#include <iostream>
#include "util_stream.cu"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int calculatePadSAD(unsigned char curr_block[BLOCK_SIZE][BLOCK_SIZE], unsigned char ref_block[SHARED_BLOCK_SIZE][SHARED_BLOCK_SIZE],
                               int motionVectorX, int motionVectorY) {
    // printf("x_idx: %d, y_idx: %d, x_ref: %d, y_ref: %d\n", x_idx, y_idx, x_ref, y_ref);
    int sad = 0;
    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        for (int j = 0; j < BLOCK_SIZE; j++)
        {
            sad += abs(curr_block[i][j] - ref_block[i + motionVectorY][j + motionVectorX]);
        }
    }
    // printf("(%d, %d), (%d, %d),sad:%d\n", x_idx, y_idx, x_ref, y_ref, sad);
    // printf("(%d, %d), (%d, %d),sad:%d\n", x_idx, y_idx, x_ref, y_ref, sad);
    return sad;
}

__global__ void constructFrame(unsigned char *d_curr_frame, unsigned char *d_ref_frame,
                          unsigned char *d_temp_frame, int *d_motion_vectors)
{
    __shared__ int s_Sad[CUDA_BLOCK_SIZE]; // 17 * 17
    __shared__ int s_mv_x[CUDA_BLOCK_SIZE];
    __shared__ int s_mv_y[CUDA_BLOCK_SIZE];

    __shared__ unsigned char ref_block[SHARED_BLOCK_SIZE][SHARED_BLOCK_SIZE]; //size = block_length + 2 * search_range = 16 + 2*8 = 32
    __shared__ unsigned char curr_block[BLOCK_SIZE][BLOCK_SIZE];

    int y_block = blockIdx.x; // row
    int x_block = blockIdx.y; // col

    int x_idx_p = x_block * BLOCK_SIZE + SEARCH_RANGE;
    int y_idx_p = y_block * BLOCK_SIZE + SEARCH_RANGE;

    int threadIdX = threadIdx.x; // 获取当前thread的X索引, row
    int threadIdY = threadIdx.y; // 获取当前thread的Y索引, col
    int currIdx = threadIdY * CUDA_BLOCK_LENGTH + threadIdX;

    int motionVectorX = (int)threadIdX; // - SEARCH_RANGE;
    int motionVectorY = (int)threadIdY; // - SEARCH_RANGE;

    //int x_ref = x_idx_p + motionVectorX;
    //int y_ref = y_idx_p + motionVectorY;

    int indXt = x_idx_p + threadIdY;
    int indYt = y_idx_p + threadIdX;

    s_mv_x[currIdx] = motionVectorY;
    s_mv_y[currIdx] = motionVectorX;

    if(threadIdX < BLOCK_SIZE && threadIdY < BLOCK_SIZE) {
        curr_block[threadIdX][threadIdY] = d_curr_frame[indYt * PADWIDTH + indXt];
        
        for(int i = 0; i < 4; i++) {
            ref_block[threadIdX + DIRECTION1_4[i][0]][threadIdY + DIRECTION1_4[i][1]] = d_ref_frame[(indYt + DIRECTION2_4[i][0]) * PADWIDTH + (indXt + DIRECTION2_4[i][1])];
        }  
    }
    __syncthreads();    

    s_Sad[currIdx] = calculatePadSAD(curr_block, ref_block, motionVectorX, motionVectorY); //

    __syncthreads();
    
    for (int stride = 2; stride < CUDA_BLOCK_SIZE; stride *= 2)
    {
        
        if (currIdx % stride == 0)
        {
            int targetIdx = currIdx + stride / 2;
            if (targetIdx < CUDA_BLOCK_SIZE && s_Sad[currIdx] > s_Sad[targetIdx])
            {
                s_Sad[currIdx] = s_Sad[targetIdx];
                s_mv_x[currIdx] = s_mv_x[targetIdx];
                s_mv_y[currIdx] = s_mv_y[targetIdx];

            }
        }
        __syncthreads();
    }    

    if (threadIdX < BLOCK_SIZE && threadIdY < BLOCK_SIZE)
    {
        

        // 确保 indXt 和 indYt 在有效范围内
        if (indXt >= 0 && indXt < PADWIDTH && indYt >= 0 && indYt < PADHEIGHT)
        {
            d_temp_frame[indYt * PADWIDTH + indXt] = ref_block[threadIdX + s_mv_y[0]][threadIdY + s_mv_x[0]];
        }
    }
}

void block_match_full_frame_stream_save_reconst(Frame* allYFrames, unsigned char* allRconstFrames) {
    //int *h_motion_vectors;

    unsigned char *d_curr_frame[STREAM_NUM];
    unsigned char *d_ref_frame[STREAM_NUM];
    unsigned char *d_temp_frame[STREAM_NUM]; // used for both prediction frame and reconst frame
    int *d_motion_vectors[STREAM_NUM];

    // int padedWidth = WIDTH + 2 * PADDING;
    // int padedHeight = HEIGHT + 2 * PADDING;

    int x_block_num = WIDTH / BLOCK_SIZE;
    int y_block_num = HEIGHT / BLOCK_SIZE;

    //size_t pixel_num = WIDTH * HEIGHT;
    size_t pixel_num = PADWIDTH * PADHEIGHT;
    size_t gop_size = GOP * pixel_num;
    //size_t frame_size = WIDTH * HEIGHT * sizeof(unsigned char);
    size_t frame_pad_size = PADWIDTH * PADHEIGHT * sizeof(unsigned char);
    size_t sad_size = x_block_num * y_block_num * sizeof(int);

    
    //h_motion_vectors = (int *)malloc(2 * sad_size);
    //h_temp_frame = (unsigned char *)malloc(frame_size);

    for (int i = 0; i < STREAM_NUM; i++) {
        hipMalloc((void **)&d_curr_frame[i], frame_pad_size);
        hipMalloc((void **)&d_ref_frame[i], frame_pad_size);
        hipMalloc((void **)&d_temp_frame[i], frame_pad_size);
        hipMemset(&d_temp_frame[i], 72, PADWIDTH * PADHEIGHT);
        hipMalloc((void **)&d_motion_vectors[i], 2 * sad_size);
    }


    // Blocks configuration
    dim3 blocks(HEIGHT / BLOCK_SIZE, WIDTH / BLOCK_SIZE);
    dim3 threadsPerBlock(CUDA_BLOCK_LENGTH, CUDA_BLOCK_LENGTH);

    // stream start
    hipStream_t stream[STREAM_NUM];
    for (int i = 0; i < STREAM_NUM; i++) {
        hipStreamCreate(&stream[i]);
    }

    for (int i = 0; i < STREAM_NUM; i++) {

        hipMemcpyAsync(d_curr_frame[i], allYFrames[0 + GOP *i].y, frame_pad_size, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(d_ref_frame[i], allYFrames[0 + GOP *i].y, frame_pad_size, hipMemcpyHostToDevice, stream[i]);
        
        // Launch the full search kernel
        constructFrame<<<blocks, threadsPerBlock, 0, stream[i]>>>(d_curr_frame[i], d_ref_frame[i], d_temp_frame[i], d_motion_vectors[i]);

        hipMemcpyAsync(allRconstFrames + gop_size * i, d_temp_frame[i], frame_pad_size, hipMemcpyDeviceToHost, stream[i]);
    }

    /**/
    for(int j = 1; j < GOP; j++) {
        for (int i = 0; i < STREAM_NUM; i++) {
            
            //if(j % 10 == 1) printf("%d %d\n", i, j);
            hipMemcpyAsync(d_curr_frame[i], allYFrames[j + GOP *i].y, frame_pad_size, hipMemcpyHostToDevice, stream[i]);
            hipMemcpyAsync(d_ref_frame[i], allYFrames[j-1 + GOP *i].y, frame_pad_size, hipMemcpyHostToDevice, stream[i]);
            //hipMemcpyAsync(d_ref_frame[i], d_temp_frame[i], frame_pad_size, hipMemcpyDeviceToDevice, stream[i]);

            constructFrame<<<blocks, threadsPerBlock, 0, stream[i]>>>(d_curr_frame[i], d_ref_frame[i], d_temp_frame[i], d_motion_vectors[i]);

            hipMemcpyAsync(allRconstFrames + pixel_num * j + gop_size * i, d_temp_frame[i], frame_pad_size, hipMemcpyDeviceToHost, stream[i]);
        }
    }
    

    //hipMemcpy(h_motion_vectors, d_motion_vectors, 2 * sad_size, hipMemcpyDeviceToHost);
    //hipMemcpy(h_temp_frame, d_temp_frame, frame_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < STREAM_NUM; i++) {
        hipFree(d_curr_frame[i]);
        hipFree(d_ref_frame[i]);
        hipFree(d_motion_vectors[i]);
        hipFree(d_temp_frame[i]);
    }

    // Free memory
}

int main()
{
    double start, end;
    start = getTimeStamp();

    Frame *allYFrames = process_yuv_frames("long_input.yuv", WIDTH, HEIGHT, FRAME_NUM);

    end = getTimeStamp();
    printf("File reading time: %f\n", end - start);
    start = getTimeStamp();
    
    //int mVSize = (int)(PADHEIGHT / BLOCK_SIZE) * (int)(PADWIDTH / BLOCK_SIZE) * 2;
    int frameSize = PADWIDTH * PADHEIGHT;
    unsigned char *allRconstFrames = (unsigned char *)malloc(FRAME_NUM * frameSize * sizeof(unsigned char));

    block_match_full_frame_stream_save_reconst(allYFrames, allRconstFrames);

    end = getTimeStamp();
    printf("Processing time: %f\n", end - start);

    /*
    FILE *recon_yFrameFile = fopen("AllReconYFrames.txt", "w"); 
    if (recon_yFrameFile == NULL) {
        printf("Error opening Y frame file.\n");
        return -1;
    }
    saveAllFrameToText(recon_yFrameFile, allRconstFrames);

    fclose(recon_yFrameFile);
    */

    hipDeviceReset();

    free(allRconstFrames);
}